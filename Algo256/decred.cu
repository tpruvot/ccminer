#include "hip/hip_runtime.h"
/**
 * Blake-256 Decred 180-Bytes input Cuda Kernel (Tested on SM 5/5.2)
 *
 * Tanguy Pruvot - Feb 2016
 *
 * Revised for optimisation by pallas @ bitcointalk - Apr 2016
 */

#include <stdint.h>
#include <memory.h>

#include <miner.h>

extern "C" {
#include <sph/sph_blake.h>
}

/* threads per block */
#define TPB 512
/* nonces per round */
#define NPR 128

/* hash by cpu with blake 256 */
extern "C" void decred_hash(void *output, const void *input)
{
	sph_blake256_context ctx;

	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 180);
	sph_blake256_close(&ctx, output);
}

#include <cuda_helper.h>

#ifdef __INTELLISENSE__
#define __byte_perm(x, y, b) x
#endif

__constant__ uint32_t _ALIGN(4) d_data[24];

/* 16 adapters max */
static uint32_t *d_resNonce[MAX_GPUS];
static uint32_t *h_resNonce[MAX_GPUS];

/* max count of found nonces in one call */
#define NBN 2
#if NBN > 1
static uint32_t extra_results[NBN] = { UINT32_MAX };
#endif


/* ############################################################################################################################### */

#define GSPREC(a,b,c,d,x,y) { \
	v[a] += (m[x] ^ c_u256[y]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a], 0, 0x1032); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
	v[a] += (m[y] ^ c_u256[x]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a], 0, 0x0321); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
}


#define GSPREC4(a0,b0,c0,d0,x0,y0,a1,b1,c1,d1,x1,y1,a2,b2,c2,d2,x2,y2,a3,b3,c3,d3,x3,y3) { \
	v[a0] += (m[x0] ^ c_u256[y0]) + v[b0]; \
	v[a1] += (m[x1] ^ c_u256[y1]) + v[b1]; \
	v[a2] += (m[x2] ^ c_u256[y2]) + v[b2]; \
	v[a3] += (m[x3] ^ c_u256[y3]) + v[b3]; \
	v[d0] = __byte_perm(v[d0] ^ v[a0], 0, 0x1032); \
	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x1032); \
	v[d2] = __byte_perm(v[d2] ^ v[a2], 0, 0x1032); \
	v[d3] = __byte_perm(v[d3] ^ v[a3], 0, 0x1032); \
	v[c0] += v[d0]; \
	v[c1] += v[d1]; \
	v[c2] += v[d2]; \
	v[c3] += v[d3]; \
	v[b0] = SPH_ROTR32(v[b0] ^ v[c0], 12); \
	v[b1] = SPH_ROTR32(v[b1] ^ v[c1], 12); \
	v[b2] = SPH_ROTR32(v[b2] ^ v[c2], 12); \
	v[b3] = SPH_ROTR32(v[b3] ^ v[c3], 12); \
	v[a0] += (m[y0] ^ c_u256[x0]) + v[b0]; \
	v[a1] += (m[y1] ^ c_u256[x1]) + v[b1]; \
	v[a2] += (m[y2] ^ c_u256[x2]) + v[b2]; \
	v[a3] += (m[y3] ^ c_u256[x3]) + v[b3]; \
	v[d0] = __byte_perm(v[d0] ^ v[a0], 0, 0x0321); \
	v[d1] = __byte_perm(v[d1] ^ v[a1], 0, 0x0321); \
	v[d2] = __byte_perm(v[d2] ^ v[a2], 0, 0x0321); \
	v[d3] = __byte_perm(v[d3] ^ v[a3], 0, 0x0321); \
	v[c0] += v[d0]; \
	v[c1] += v[d1]; \
	v[c2] += v[d2]; \
	v[c3] += v[d3]; \
	v[b0] = SPH_ROTR32(v[b0] ^ v[c0], 7); \
	v[b1] = SPH_ROTR32(v[b1] ^ v[c1], 7); \
	v[b2] = SPH_ROTR32(v[b2] ^ v[c2], 7); \
	v[b3] = SPH_ROTR32(v[b3] ^ v[c3], 7); \
}


static const __constant__ uint32_t c_u256[16] = {
	0x243F6A88, 0x85A308D3, 0x13198A2E, 0x03707344,
	0xA4093822, 0x299F31D0, 0x082EFA98, 0xEC4E6C89,
	0x452821E6, 0x38D01377, 0xBE5466CF, 0x34E90C6C,
	0xC0AC29B7, 0xC97C50DD, 0x3F84D5B5, 0xB5470917
};


__device__ __forceinline__
uint32_t blake256_compress_14(uint32_t *m, uint32_t *v_init, uint32_t d_data6, uint32_t d_data7)
{
	uint32_t v[16];

	#pragma unroll
	for (uint32_t i = 0; i < 16; i++) v[i] = v_init[i];
	// these two are not modified:
	v[ 9] = 0x85A308D3;
	v[13] = 0x299F31D0 ^ (180U*8U);

	// round 1 with nonce
	GSPREC(1, 5, 0x9, 0xD, 2,  3);
	GSPREC(0, 5, 0xA, 0xF, 8,  9);
	GSPREC(1, 6, 0xB, 0xC, 10, 11);
	GSPREC(2, 7, 0x8, 0xD, 12, 13);
	GSPREC(3, 4, 0x9, 0xE, 14, 15);
	// round 2
	GSPREC4(0, 4, 0x8, 0xC, 14, 10, 1, 5, 0x9, 0xD, 4,  8, 2, 6, 0xA, 0xE, 9,  15, 3, 7, 0xB, 0xF, 13, 6);
	GSPREC4(0, 5, 0xA, 0xF, 1,  12, 1, 6, 0xB, 0xC, 0,  2, 2, 7, 0x8, 0xD, 11, 7, 3, 4, 0x9, 0xE, 5,  3);
	// round 3
	GSPREC4(0, 4, 0x8, 0xC, 11, 8, 1, 5, 0x9, 0xD, 12, 0, 2, 6, 0xA, 0xE, 5,  2, 3, 7, 0xB, 0xF, 15, 13);
	GSPREC4(0, 5, 0xA, 0xF, 10, 14, 1, 6, 0xB, 0xC, 3,  6, 2, 7, 0x8, 0xD, 7,  1, 3, 4, 0x9, 0xE, 9,  4);
	// round 4
	GSPREC4(0, 4, 0x8, 0xC, 7,  9, 1, 5, 0x9, 0xD, 3,  1, 2, 6, 0xA, 0xE, 13, 12, 3, 7, 0xB, 0xF, 11, 14);
	GSPREC4(0, 5, 0xA, 0xF, 2,  6, 1, 6, 0xB, 0xC, 5,  10, 2, 7, 0x8, 0xD, 4,  0, 3, 4, 0x9, 0xE, 15, 8);
	// round 5
	GSPREC4(0, 4, 0x8, 0xC, 9,  0, 1, 5, 0x9, 0xD, 5,  7, 2, 6, 0xA, 0xE, 2,  4, 3, 7, 0xB, 0xF, 10, 15);
	GSPREC4(0, 5, 0xA, 0xF, 14, 1, 1, 6, 0xB, 0xC, 11, 12, 2, 7, 0x8, 0xD, 6,  8, 3, 4, 0x9, 0xE, 3,  13);
	// round 6
	GSPREC4(0, 4, 0x8, 0xC, 2, 12, 1, 5, 0x9, 0xD, 6, 10, 2, 6, 0xA, 0xE, 0, 11, 3, 7, 0xB, 0xF, 8, 3);
	GSPREC4(0, 5, 0xA, 0xF, 4, 13, 1, 6, 0xB, 0xC, 7, 5, 2, 7, 0x8, 0xD, 15,14, 3, 4, 0x9, 0xE, 1, 9);
	// round 7
	GSPREC4(0, 4, 0x8, 0xC, 12, 5, 1, 5, 0x9, 0xD, 1, 15, 2, 6, 0xA, 0xE, 14,13, 3, 7, 0xB, 0xF, 4, 10);
	GSPREC4(0, 5, 0xA, 0xF, 0,  7, 1, 6, 0xB, 0xC, 6,  3, 2, 7, 0x8, 0xD, 9,  2, 3, 4, 0x9, 0xE, 8, 11);
	/*
	// round 8
	GSPREC4(0, 4, 0x8, 0xC, 13,11, 1, 5, 0x9, 0xD, 7, 14, 2, 6, 0xA, 0xE, 12, 1, 3, 7, 0xB, 0xF, 3,  9);
	GSPREC4(0, 5, 0xA, 0xF, 5,  0, 1, 6, 0xB, 0xC, 15, 4, 2, 7, 0x8, 0xD, 8,  6, 3, 4, 0x9, 0xE, 2, 10);
	// round 9
	GSPREC4(0, 4, 0x8, 0xC, 6, 15, 1, 5, 0x9, 0xD, 14, 9, 2, 6, 0xA, 0xE, 11, 3, 3, 7, 0xB, 0xF, 0,  8);
	GSPREC4(0, 5, 0xA, 0xF, 12, 2, 1, 6, 0xB, 0xC, 13, 7, 2, 7, 0x8, 0xD, 1,  4, 3, 4, 0x9, 0xE, 10, 5);
	// round 10
	GSPREC4(0, 4, 0x8, 0xC, 10, 2, 1, 5, 0x9, 0xD, 8,  4, 2, 6, 0xA, 0xE, 7,  6, 3, 7, 0xB, 0xF, 1,  5);
	GSPREC4(0, 5, 0xA, 0xF, 15,11, 1, 6, 0xB, 0xC, 9, 14, 2, 7, 0x8, 0xD, 3, 12, 3, 4, 0x9, 0xE, 13, 0);
	// round 11
	GSPREC4(0, 4, 0x8, 0xC, 0,  1, 1, 5, 0x9, 0xD, 2,  3, 2, 6, 0xA, 0xE, 4,  5, 3, 7, 0xB, 0xF, 6,  7);
	GSPREC4(0, 5, 0xA, 0xF, 8,  9, 1, 6, 0xB, 0xC, 10,11, 2, 7, 0x8, 0xD, 12,13, 3, 4, 0x9, 0xE, 14,15);
	// round 12
	GSPREC4(0, 4, 0x8, 0xC, 14,10, 1, 5, 0x9, 0xD, 4,  8, 2, 6, 0xA, 0xE, 9, 15, 3, 7, 0xB, 0xF, 13, 6);
	GSPREC4(0, 5, 0xA, 0xF, 1, 12, 1, 6, 0xB, 0xC, 0,  2, 2, 7, 0x8, 0xD, 11, 7, 3, 4, 0x9, 0xE, 5,  3);
	// round 13
	GSPREC4(0, 4, 0x8, 0xC, 11, 8, 1, 5, 0x9, 0xD, 12, 0, 2, 6, 0xA, 0xE, 5,  2, 3, 7, 0xB, 0xF, 15,13);
	GSPREC4(0, 5, 0xA, 0xF, 10,14, 1, 6, 0xB, 0xC, 3,  6, 2, 7, 0x8, 0xD, 7,  1, 3, 4, 0x9, 0xE, 9,  4);
	*/
	// round 8
	GSPREC(0, 4, 0x8, 0xC, 13,11);
	GSPREC(1, 5, 0x9, 0xD, 7, 14);
	GSPREC(2, 6, 0xA, 0xE, 12, 1);
	GSPREC(3, 7, 0xB, 0xF, 3,  9);
	GSPREC(0, 5, 0xA, 0xF, 5,  0);
	GSPREC(1, 6, 0xB, 0xC, 15, 4);
	GSPREC(2, 7, 0x8, 0xD, 8,  6);
	GSPREC(3, 4, 0x9, 0xE, 2, 10);
	// round 9
	GSPREC(0, 4, 0x8, 0xC, 6, 15);
	GSPREC(1, 5, 0x9, 0xD, 14, 9);
	GSPREC(2, 6, 0xA, 0xE, 11, 3);
	GSPREC(3, 7, 0xB, 0xF, 0,  8);
	GSPREC(0, 5, 0xA, 0xF, 12, 2);
	GSPREC(1, 6, 0xB, 0xC, 13, 7);
	GSPREC(2, 7, 0x8, 0xD, 1,  4);
	GSPREC(3, 4, 0x9, 0xE, 10, 5);
	// round 10
	GSPREC(0, 4, 0x8, 0xC, 10, 2);
	GSPREC(1, 5, 0x9, 0xD, 8,  4);
	GSPREC(2, 6, 0xA, 0xE, 7,  6);
	GSPREC(3, 7, 0xB, 0xF, 1,  5);
	GSPREC(0, 5, 0xA, 0xF, 15,11);
	GSPREC(1, 6, 0xB, 0xC, 9, 14);
	GSPREC(2, 7, 0x8, 0xD, 3, 12);
	GSPREC(3, 4, 0x9, 0xE, 13, 0);
	// round 11
	GSPREC(0, 4, 0x8, 0xC, 0,  1);
	GSPREC(1, 5, 0x9, 0xD, 2,  3);
	GSPREC(2, 6, 0xA, 0xE, 4,  5);
	GSPREC(3, 7, 0xB, 0xF, 6,  7);
	GSPREC(0, 5, 0xA, 0xF, 8,  9);
	GSPREC(1, 6, 0xB, 0xC, 10,11);
	GSPREC(2, 7, 0x8, 0xD, 12,13);
	GSPREC(3, 4, 0x9, 0xE, 14,15);
	// round 12
	GSPREC(0, 4, 0x8, 0xC, 14,10);
	GSPREC(1, 5, 0x9, 0xD, 4,  8);
	GSPREC(2, 6, 0xA, 0xE, 9, 15);
	GSPREC(3, 7, 0xB, 0xF, 13, 6);
	GSPREC(0, 5, 0xA, 0xF, 1, 12);
	GSPREC(1, 6, 0xB, 0xC, 0,  2);
	GSPREC(2, 7, 0x8, 0xD, 11, 7);
	GSPREC(3, 4, 0x9, 0xE, 5,  3);
	// round 13
	GSPREC(0, 4, 0x8, 0xC, 11, 8);
	GSPREC(1, 5, 0x9, 0xD, 12, 0);
	GSPREC(2, 6, 0xA, 0xE, 5,  2);
	GSPREC(3, 7, 0xB, 0xF, 15,13);
	GSPREC(0, 5, 0xA, 0xF, 10,14);
	GSPREC(1, 6, 0xB, 0xC, 3,  6);
	GSPREC(2, 7, 0x8, 0xD, 7,  1);
	GSPREC(3, 4, 0x9, 0xE, 9,  4);
	// round 14
	GSPREC(0, 4, 0x8, 0xC, 7,  9);
	GSPREC(1, 5, 0x9, 0xD, 3,  1);
	GSPREC(2, 6, 0xA, 0xE, 13,12);
	GSPREC(3, 7, 0xB, 0xF, 11,14);
	GSPREC(0, 5, 0xA, 0xF, 2,  6);
	GSPREC(2, 7, 0x8, 0xD, 4,  0);

	if ((d_data7 ^ v[7] ^ v[15]) == 0) {
		GSPREC(1, 6, 0xB, 0xC, 5, 10);
		GSPREC(3, 4, 0x9, 0xE, 15, 8);
		return (d_data6 ^ v[6] ^ v[14]);
	}
	return UINT32_MAX;
}


/* ############################################################################################################################### */

// ------ Close: Last 52/64 bytes ------
__global__
void blake256_gpu_hash_nonce(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce, const uint64_t highTarget)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
//	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread * NPR;
		uint32_t m[16], v[16], temp;
		const uint32_t d_data6 = d_data[6], d_data7 = d_data[7];

		#pragma unroll
		for(int i = 0; i < 8; i++) v[i] = d_data[i];

		#pragma unroll
		for (uint32_t i = 0; i < 16; i++) m[i] = d_data[i+8U];

		v[ 8] = 0x243F6A88;
		v[ 9] = 0x85A308D3;
		v[10] = 0x13198A2E;
		v[11] = 0x03707344;

		v[12] = 0xA4093822 ^ (180U*8U);
		v[13] = 0x299F31D0 ^ (180U*8U);
		v[14] = 0x082EFA98;
		v[15] = 0xEC4E6C89;

		// round 1 without nonce
		GSPREC(0, 4, 0x8, 0xC, 0,  1);
		GSPREC(2, 6, 0xA, 0xE, 4,  5);
		GSPREC(3, 7, 0xB, 0xF, 6,  7);

		for (m[3] = nonce; m[3] < nonce + NPR; m[3]++) {
			temp = blake256_compress_14(m, v, d_data6, d_data7);

			if (temp != UINT32_MAX && cuda_swab32(temp) <= highTarget) {
				#if NBN == 2
					if (resNonce[0] != UINT32_MAX) resNonce[1] = m[3];
					else resNonce[0] = m[3];
				#else
					resNonce[0] = m[3];
				#endif
			}
		}
	}
}


__host__
static uint32_t decred_cpu_hash_nonce(const int thr_id, const uint32_t threads, const uint32_t startNonce, const uint64_t highTarget)
{
	uint32_t result = UINT32_MAX;
	const uint32_t real_threads = threads / NPR;

	dim3 grid((real_threads + TPB-1)/TPB);
	dim3 block(TPB);

	/* Check error on Ctrl+C or kill to prevent segfaults on exit */
	if (hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t)) != hipSuccess)
		return result;

	blake256_gpu_hash_nonce <<<grid, block>>> (real_threads, startNonce, d_resNonce[thr_id], highTarget);
	hipDeviceSynchronize();

	if (hipSuccess == hipMemcpy(h_resNonce[thr_id], d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost)) {
		result = h_resNonce[thr_id][0];
#if NBN > 1
		for (int n=0; n < (NBN-1); n++)
			extra_results[n] = h_resNonce[thr_id][n+1];
#endif
	}
	return result;
}


__host__
static void decred_midstate_128(uint32_t *output, const uint32_t *input)
{
	sph_blake256_context ctx;

	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 128);

	memcpy(output, (void*)ctx.H, 32);
}


__host__
void decred_cpu_setBlock_52(uint32_t *penddata, const uint32_t *midstate, const uint32_t *ptarget)
{
	uint32_t _ALIGN(64) data[24];
	memcpy(data, midstate, 32);
	// pre swab32
	for (int i=0; i<13; i++)
		data[8+i] = swab32(penddata[i]);
	data[21] = 0x80000001;
	data[22] = 0;
	data[23] = 0x000005a0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_data), data, 32 + 64, 0, hipMemcpyHostToDevice));
}


/* ############################################################################################################################### */

static bool init[MAX_GPUS] = { 0 };

// nonce position is different in decred
#define DCR_NONCE_OFT32 35

extern "C" int scanhash_decred(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[48];
	uint32_t _ALIGN(64) midstate[8];

	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	uint32_t *pnonce = &pdata[DCR_NONCE_OFT32];

	const uint32_t first_nonce = *pnonce;
	uint64_t targetHigh = ((uint64_t*)ptarget)[3];

	int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 29 : 25;
	if (device_sm[dev_id] < 350) intensity = 22;

	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	int rc = 0;

	if (opt_benchmark) {
		targetHigh = 0x1ULL << 32;
		ptarget[6] = swab32(0xff);
	}

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}

		CUDA_CALL_OR_RET_X(hipMalloc(&d_resNonce[thr_id], NBN * sizeof(uint32_t)), -1);
		CUDA_CALL_OR_RET_X(hipHostMalloc(&h_resNonce[thr_id], NBN * sizeof(uint32_t)), -1);
		init[thr_id] = true;
	}

	memcpy(endiandata, pdata, 180);
	decred_midstate_128(midstate, endiandata);
	decred_cpu_setBlock_52(&pdata[32], midstate, ptarget);

	do {
		// GPU HASH
		uint32_t foundNonce = decred_cpu_hash_nonce(thr_id, throughput, (*pnonce), targetHigh);

		if (foundNonce != UINT32_MAX)
		{
			uint32_t vhashcpu[8];
			uint32_t Htarg = ptarget[6];

			be32enc(&endiandata[DCR_NONCE_OFT32], foundNonce);
			decred_hash(vhashcpu, endiandata);

			if (vhashcpu[6] <= Htarg && fulltest(vhashcpu, ptarget))
			{
				rc = 1;
				work_set_target_ratio(work, vhashcpu);
				*hashes_done = (*pnonce) - first_nonce + throughput;
				work->nonces[0] = swab32(foundNonce);
#if NBN > 1
				if (extra_results[0] != UINT32_MAX) {
					be32enc(&endiandata[DCR_NONCE_OFT32], extra_results[0]);
					decred_hash(vhashcpu, endiandata);
					if (vhashcpu[6] <= Htarg && fulltest(vhashcpu, ptarget)) {
						work->nonces[1] = swab32(extra_results[0]);
						if (bn_hash_target_ratio(vhashcpu, ptarget) > work->shareratio) {
							work_set_target_ratio(work, vhashcpu);
							xchg(work->nonces[1], work->nonces[0]);
						}
						rc = 2;
					}
					extra_results[0] = UINT32_MAX;
				}
#endif
				*pnonce = work->nonces[0];
				return rc;
			}
			else if (opt_debug) {
				applog_hash(ptarget);
				applog_compare_hash(vhashcpu, ptarget);
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}

		*pnonce += throughput;

	} while (!work_restart[thr_id].restart && max_nonce > (uint64_t)throughput + (*pnonce));

	*hashes_done = (*pnonce) - first_nonce;
	return rc;
}


// cleanup
extern "C" void free_decred(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipHostFree(h_resNonce[thr_id]);
	hipFree(d_resNonce[thr_id]);

	init[thr_id] = false;

	hipDeviceSynchronize();
}
