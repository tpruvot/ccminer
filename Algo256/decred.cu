#include "hip/hip_runtime.h"
/**
 * Blake-256 Decred 180-Bytes input Cuda Kernel (Tested on SM 5/5.2)
 *
 * Tanguy Pruvot - Feb 2016
 */

#include <stdint.h>
#include <memory.h>

#include <miner.h>

extern "C" {
#include <sph/sph_blake.h>
}

/* threads per block */
#define TPB 256

/* hash by cpu with blake 256 */
extern "C" void decred_hash(void *output, const void *input)
{
	sph_blake256_context ctx;

	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 180);
	sph_blake256_close(&ctx, output);
}

#include <cuda_helper.h>

#ifdef __INTELLISENSE__
#define __byte_perm(x, y, b) x
#endif

__constant__ uint32_t _ALIGN(4) d_data[24];

/* 16 adapters max */
static uint32_t *d_resNonce[MAX_GPUS];
static uint32_t *h_resNonce[MAX_GPUS];

/* max count of found nonces in one call */
#define NBN 2
#if NBN > 1
static uint32_t extra_results[NBN] = { UINT32_MAX };
#endif

/* ############################################################################################################################### */

#define GSPREC(a,b,c,d,x,y) { \
	v[a] += (m[x] ^ c_u256[y]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a], 0, 0x1032); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 12); \
	v[a] += (m[y] ^ c_u256[x]) + v[b]; \
	v[d] = __byte_perm(v[d] ^ v[a], 0, 0x0321); \
	v[c] += v[d]; \
	v[b] = SPH_ROTR32(v[b] ^ v[c], 7); \
}

__device__ __forceinline__
void blake256_compress_14(uint32_t *h, const uint32_t nonce, const uint32_t T0)
{
	uint32_t v[16];

	#pragma unroll 8
	for(uint32_t i = 0; i < 8; i++)
		v[i] = h[i];

	const uint32_t c_u256[16] = {
		0x243F6A88, 0x85A308D3, 0x13198A2E, 0x03707344,
		0xA4093822, 0x299F31D0, 0x082EFA98, 0xEC4E6C89,
		0x452821E6, 0x38D01377, 0xBE5466CF, 0x34E90C6C,
		0xC0AC29B7, 0xC97C50DD, 0x3F84D5B5, 0xB5470917
	};

	v[ 8] = c_u256[0];
	v[ 9] = c_u256[1];
	v[10] = c_u256[2];
	v[11] = c_u256[3];

	v[12] = c_u256[4] ^ T0;
	v[13] = c_u256[5] ^ T0;
	v[14] = c_u256[6];
	v[15] = c_u256[7];

	uint32_t m[16];

	m[0] = d_data[8];
	m[1] = d_data[9];
	m[2] = d_data[10];
	m[3] = nonce;

	#pragma unroll
	for (uint32_t i = 4; i < 16; i++) {
		m[i] = d_data[i+8U];
	}

	// round 1
	GSPREC(0, 4, 0x8, 0xC, 0,  1);
	GSPREC(1, 5, 0x9, 0xD, 2,  3);
	GSPREC(2, 6, 0xA, 0xE, 4,  5);
	GSPREC(3, 7, 0xB, 0xF, 6,  7);
	GSPREC(0, 5, 0xA, 0xF, 8,  9);
	GSPREC(1, 6, 0xB, 0xC, 10, 11);
	GSPREC(2, 7, 0x8, 0xD, 12, 13);
	GSPREC(3, 4, 0x9, 0xE, 14, 15);
	// round 2
	GSPREC(0, 4, 0x8, 0xC, 14, 10);
	GSPREC(1, 5, 0x9, 0xD, 4,  8);
	GSPREC(2, 6, 0xA, 0xE, 9,  15);
	GSPREC(3, 7, 0xB, 0xF, 13, 6);
	GSPREC(0, 5, 0xA, 0xF, 1,  12);
	GSPREC(1, 6, 0xB, 0xC, 0,  2);
	GSPREC(2, 7, 0x8, 0xD, 11, 7);
	GSPREC(3, 4, 0x9, 0xE, 5,  3);
	// round 3
	GSPREC(0, 4, 0x8, 0xC, 11, 8);
	GSPREC(1, 5, 0x9, 0xD, 12, 0);
	GSPREC(2, 6, 0xA, 0xE, 5,  2);
	GSPREC(3, 7, 0xB, 0xF, 15, 13);
	GSPREC(0, 5, 0xA, 0xF, 10, 14);
	GSPREC(1, 6, 0xB, 0xC, 3,  6);
	GSPREC(2, 7, 0x8, 0xD, 7,  1);
	GSPREC(3, 4, 0x9, 0xE, 9,  4);
	// round 4
	GSPREC(0, 4, 0x8, 0xC, 7,  9);
	GSPREC(1, 5, 0x9, 0xD, 3,  1);
	GSPREC(2, 6, 0xA, 0xE, 13, 12);
	GSPREC(3, 7, 0xB, 0xF, 11, 14);
	GSPREC(0, 5, 0xA, 0xF, 2,  6);
	GSPREC(1, 6, 0xB, 0xC, 5,  10);
	GSPREC(2, 7, 0x8, 0xD, 4,  0);
	GSPREC(3, 4, 0x9, 0xE, 15, 8);
	// round 5
	GSPREC(0, 4, 0x8, 0xC, 9,  0);
	GSPREC(1, 5, 0x9, 0xD, 5,  7);
	GSPREC(2, 6, 0xA, 0xE, 2,  4);
	GSPREC(3, 7, 0xB, 0xF, 10, 15);
	GSPREC(0, 5, 0xA, 0xF, 14, 1);
	GSPREC(1, 6, 0xB, 0xC, 11, 12);
	GSPREC(2, 7, 0x8, 0xD, 6,  8);
	GSPREC(3, 4, 0x9, 0xE, 3,  13);
	// round 6
	GSPREC(0, 4, 0x8, 0xC, 2, 12);
	GSPREC(1, 5, 0x9, 0xD, 6, 10);
	GSPREC(2, 6, 0xA, 0xE, 0, 11);
	GSPREC(3, 7, 0xB, 0xF, 8, 3);
	GSPREC(0, 5, 0xA, 0xF, 4, 13);
	GSPREC(1, 6, 0xB, 0xC, 7, 5);
	GSPREC(2, 7, 0x8, 0xD, 15,14);
	GSPREC(3, 4, 0x9, 0xE, 1, 9);
	// round 7
	GSPREC(0, 4, 0x8, 0xC, 12, 5);
	GSPREC(1, 5, 0x9, 0xD, 1, 15);
	GSPREC(2, 6, 0xA, 0xE, 14,13);
	GSPREC(3, 7, 0xB, 0xF, 4, 10);
	GSPREC(0, 5, 0xA, 0xF, 0,  7);
	GSPREC(1, 6, 0xB, 0xC, 6,  3);
	GSPREC(2, 7, 0x8, 0xD, 9,  2);
	GSPREC(3, 4, 0x9, 0xE, 8, 11);
	// round 8
	GSPREC(0, 4, 0x8, 0xC, 13,11);
	GSPREC(1, 5, 0x9, 0xD, 7, 14);
	GSPREC(2, 6, 0xA, 0xE, 12, 1);
	GSPREC(3, 7, 0xB, 0xF, 3,  9);
	GSPREC(0, 5, 0xA, 0xF, 5,  0);
	GSPREC(1, 6, 0xB, 0xC, 15, 4);
	GSPREC(2, 7, 0x8, 0xD, 8,  6);
	GSPREC(3, 4, 0x9, 0xE, 2, 10);
	// round 9
	GSPREC(0, 4, 0x8, 0xC, 6, 15);
	GSPREC(1, 5, 0x9, 0xD, 14, 9);
	GSPREC(2, 6, 0xA, 0xE, 11, 3);
	GSPREC(3, 7, 0xB, 0xF, 0,  8);
	GSPREC(0, 5, 0xA, 0xF, 12, 2);
	GSPREC(1, 6, 0xB, 0xC, 13, 7);
	GSPREC(2, 7, 0x8, 0xD, 1,  4);
	GSPREC(3, 4, 0x9, 0xE, 10, 5);
	// round 10
	GSPREC(0, 4, 0x8, 0xC, 10, 2);
	GSPREC(1, 5, 0x9, 0xD, 8,  4);
	GSPREC(2, 6, 0xA, 0xE, 7,  6);
	GSPREC(3, 7, 0xB, 0xF, 1,  5);
	GSPREC(0, 5, 0xA, 0xF, 15,11);
	GSPREC(1, 6, 0xB, 0xC, 9, 14);
	GSPREC(2, 7, 0x8, 0xD, 3, 12);
	GSPREC(3, 4, 0x9, 0xE, 13, 0);
	// round 11
	GSPREC(0, 4, 0x8, 0xC, 0,  1);
	GSPREC(1, 5, 0x9, 0xD, 2,  3);
	GSPREC(2, 6, 0xA, 0xE, 4,  5);
	GSPREC(3, 7, 0xB, 0xF, 6,  7);
	GSPREC(0, 5, 0xA, 0xF, 8,  9);
	GSPREC(1, 6, 0xB, 0xC, 10,11);
	GSPREC(2, 7, 0x8, 0xD, 12,13);
	GSPREC(3, 4, 0x9, 0xE, 14,15);
	// round 12
	GSPREC(0, 4, 0x8, 0xC, 14,10);
	GSPREC(1, 5, 0x9, 0xD, 4,  8);
	GSPREC(2, 6, 0xA, 0xE, 9, 15);
	GSPREC(3, 7, 0xB, 0xF, 13, 6);
	GSPREC(0, 5, 0xA, 0xF, 1, 12);
	GSPREC(1, 6, 0xB, 0xC, 0,  2);
	GSPREC(2, 7, 0x8, 0xD, 11, 7);
	GSPREC(3, 4, 0x9, 0xE, 5,  3);
	// round 13
	GSPREC(0, 4, 0x8, 0xC, 11, 8);
	GSPREC(1, 5, 0x9, 0xD, 12, 0);
	GSPREC(2, 6, 0xA, 0xE, 5,  2);
	GSPREC(3, 7, 0xB, 0xF, 15,13);
	GSPREC(0, 5, 0xA, 0xF, 10,14);
	GSPREC(1, 6, 0xB, 0xC, 3,  6);
	GSPREC(2, 7, 0x8, 0xD, 7,  1);
	GSPREC(3, 4, 0x9, 0xE, 9,  4);
	// round 14
	GSPREC(0, 4, 0x8, 0xC, 7,  9);
	GSPREC(1, 5, 0x9, 0xD, 3,  1);
	GSPREC(2, 6, 0xA, 0xE, 13,12);
	GSPREC(3, 7, 0xB, 0xF, 11,14);
	GSPREC(0, 5, 0xA, 0xF, 2,  6);
	GSPREC(2, 7, 0x8, 0xD, 4,  0);

	h[7] ^= v[7] ^ v[15];

	if (h[7] == 0) {
		GSPREC(1, 6, 0xB, 0xC, 5, 10);
		GSPREC(3, 4, 0x9, 0xE, 15, 8);
		h[6] ^= v[6] ^ v[14];
	}
}

/* ############################################################################################################################### */

__global__
void blake256_gpu_hash_nonce(const uint32_t threads, const uint32_t startNonce, uint32_t *resNonce, const uint64_t highTarget)
{
	uint32_t thread = (blockDim.x * blockIdx.x + threadIdx.x);
//	if (thread < threads)
	{
		const uint32_t nonce = startNonce + thread;
		uint32_t h[8];

		#pragma unroll
		for(int i=0; i < 8; i++) {
			h[i] = d_data[i];
		}

		// ------ Close: Last 52/64 bytes ------

		blake256_compress_14(h, nonce, (180U*8U));

		if (h[7] == 0 && cuda_swab32(h[6]) <= highTarget) {
#if NBN == 2
			if (resNonce[0] != UINT32_MAX)
				resNonce[1] = nonce;
			else
				resNonce[0] = nonce;
#else
			resNonce[0] = nonce;
#endif
		}
	}
}

__host__
static uint32_t decred_cpu_hash_nonce(const int thr_id, const uint32_t threads, const uint32_t startNonce, const uint64_t highTarget)
{
	uint32_t result = UINT32_MAX;

	dim3 grid((threads + TPB-1)/TPB);
	dim3 block(TPB);

	/* Check error on Ctrl+C or kill to prevent segfaults on exit */
	if (hipMemset(d_resNonce[thr_id], 0xff, NBN*sizeof(uint32_t)) != hipSuccess)
		return result;

	blake256_gpu_hash_nonce <<<grid, block>>> (threads, startNonce, d_resNonce[thr_id], highTarget);
	hipDeviceSynchronize();

	if (hipSuccess == hipMemcpy(h_resNonce[thr_id], d_resNonce[thr_id], NBN*sizeof(uint32_t), hipMemcpyDeviceToHost)) {
		result = h_resNonce[thr_id][0];
#if NBN > 1
		for (int n=0; n < (NBN-1); n++)
			extra_results[n] = h_resNonce[thr_id][n+1];
#endif
	}
	return result;
}

__host__
static void decred_midstate_128(uint32_t *output, const uint32_t *input)
{
	sph_blake256_context ctx;

	sph_blake256_set_rounds(14);

	sph_blake256_init(&ctx);
	sph_blake256(&ctx, input, 128);

	memcpy(output, (void*)ctx.H, 32);
}

__host__
void decred_cpu_setBlock_52(uint32_t *penddata, const uint32_t *midstate, const uint32_t *ptarget)
{
	uint32_t _ALIGN(64) data[24];
	memcpy(data, midstate, 32);
	// pre swab32
	for (int i=0; i<13; i++)
		data[8+i] = swab32(penddata[i]);
	data[21] = 0x80000001;
	data[22] = 0;
	data[23] = 0x000005a0;
	CUDA_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(d_data), data, 32 + 64, 0, hipMemcpyHostToDevice));
}

/* ############################################################################################################################### */

static bool init[MAX_GPUS] = { 0 };

// nonce position is different in decred
#define DCR_NONCE_OFT32 35

extern "C" int scanhash_decred(int thr_id, struct work* work, uint32_t max_nonce, unsigned long *hashes_done)
{
	uint32_t _ALIGN(64) endiandata[48];
	uint32_t _ALIGN(64) midstate[8];

	uint32_t *pdata = work->data;
	uint32_t *ptarget = work->target;
	uint32_t *pnonce = &pdata[DCR_NONCE_OFT32];

	const uint32_t first_nonce = *pnonce;
	uint64_t targetHigh = ((uint64_t*)ptarget)[3];

	int dev_id = device_map[thr_id];
	int intensity = (device_sm[dev_id] > 500 && !is_windows()) ? 29 : 25;
	if (device_sm[dev_id] < 350) intensity = 22;

	uint32_t throughput = cuda_default_throughput(thr_id, 1U << intensity);
	if (init[thr_id]) throughput = min(throughput, max_nonce - first_nonce);

	int rc = 0;

	if (opt_benchmark) {
		targetHigh = 0x1ULL << 32;
		ptarget[6] = swab32(0xff);
	}

	if (!init[thr_id])
	{
		hipSetDevice(dev_id);
		if (opt_cudaschedule == -1 && gpu_threads == 1) {
			hipDeviceReset();
			// reduce cpu usage (linux)
			hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
			hipDeviceSetCacheConfig(hipFuncCachePreferL1);
			CUDA_LOG_ERROR();
		}

		CUDA_CALL_OR_RET_X(hipMalloc(&d_resNonce[thr_id], NBN * sizeof(uint32_t)), -1);
		CUDA_CALL_OR_RET_X(hipHostMalloc(&h_resNonce[thr_id], NBN * sizeof(uint32_t)), -1);
		init[thr_id] = true;
	}

	memcpy(endiandata, pdata, 180);
	decred_midstate_128(midstate, endiandata);
	decred_cpu_setBlock_52(&pdata[32], midstate, ptarget);

	do {
		// GPU HASH
		uint32_t foundNonce = decred_cpu_hash_nonce(thr_id, throughput, (*pnonce), targetHigh);

		if (foundNonce != UINT32_MAX)
		{
			uint32_t vhashcpu[8];
			uint32_t Htarg = ptarget[6];

			be32enc(&endiandata[DCR_NONCE_OFT32], foundNonce);
			decred_hash(vhashcpu, endiandata);

			if (vhashcpu[6] <= Htarg && fulltest(vhashcpu, ptarget))
			{
				rc = 1;
				work_set_target_ratio(work, vhashcpu);
				*hashes_done = (*pnonce) - first_nonce + throughput;
				work->nonces[0] = swab32(foundNonce);
#if NBN > 1
				if (extra_results[0] != UINT32_MAX) {
					be32enc(&endiandata[DCR_NONCE_OFT32], extra_results[0]);
					decred_hash(vhashcpu, endiandata);
					if (vhashcpu[6] <= Htarg && fulltest(vhashcpu, ptarget)) {
						work->nonces[1] = swab32(extra_results[0]);
						if (bn_hash_target_ratio(vhashcpu, ptarget) > work->shareratio) {
							work_set_target_ratio(work, vhashcpu);
							xchg(work->nonces[1], work->nonces[0]);
						}
						rc = 2;
					}
					extra_results[0] = UINT32_MAX;
				}
#endif
				*pnonce = work->nonces[0];
				return rc;
			}
			else if (opt_debug) {
				applog_hash(ptarget);
				applog_compare_hash(vhashcpu, ptarget);
				gpulog(LOG_WARNING, thr_id, "result for %08x does not validate on CPU!", foundNonce);
			}
		}

		*pnonce += throughput;

	} while (!work_restart[thr_id].restart && max_nonce > (uint64_t)throughput + (*pnonce));

	*hashes_done = (*pnonce) - first_nonce;
	return rc;
}

// cleanup
extern "C" void free_decred(int thr_id)
{
	if (!init[thr_id])
		return;

	hipDeviceSynchronize();

	hipHostFree(h_resNonce[thr_id]);
	hipFree(d_resNonce[thr_id]);

	init[thr_id] = false;

	hipDeviceSynchronize();
}

